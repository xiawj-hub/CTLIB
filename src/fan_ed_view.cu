#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 256
#define GRID_DIM 512

template <typename scalar_t>
__device__ scalar_t map_x(scalar_t sourcex, scalar_t sourcey, scalar_t detx, scalar_t dety) {
    return (sourcex * dety - sourcey * detx) / (dety - sourcey);
}

template <typename scalar_t>
__device__ scalar_t map_y(scalar_t sourcex, scalar_t sourcey, scalar_t detx, scalar_t dety) {
    return (sourcey * detx - sourcex * dety) / (detx - sourcex);
}

template <typename scalar_t>
__device__ scalar_t cweight(scalar_t sourcex, scalar_t sourcey, scalar_t detx, scalar_t dety, scalar_t r) {
    scalar_t d = (sourcex * detx + sourcey * dety) / r;
    return r * r / ((r - d) * (r - d));
}

template <typename scalar_t>
__global__ void prj_fan_ed(
    const torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> image,
    torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> projection,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> prj_views,
    const scalar_t* __restrict__ views, const scalar_t* __restrict__ dets,
    const scalar_t* __restrict__ width, const scalar_t* __restrict__ height,
    const scalar_t* __restrict__ dImg, const scalar_t* __restrict__ dDet,
    const scalar_t* __restrict__ dAng, const scalar_t* __restrict__ s2r,
    const scalar_t* __restrict__ d2r, const scalar_t* __restrict__ binshift) {
    
    __shared__ unsigned int nblocks;
    __shared__ unsigned int idxchannel;
    __shared__ unsigned int idxview;
    nblocks = ceil(*views / gridDim.y);
    idxchannel = blockIdx.x % nblocks;
    idxview = idxchannel * gridDim.y + blockIdx.y;
    if (idxview >= *views)   return;
    idxchannel = blockIdx.x / nblocks;
    __shared__ scalar_t prj[BLOCK_DIM];
    __shared__ scalar_t dPoint[BLOCK_DIM];
    __shared__ scalar_t coef[BLOCK_DIM];
    __shared__ scalar_t dImage;    
    __shared__ scalar_t sourcex;
    __shared__ scalar_t sourcey;
    __shared__ scalar_t dcx;
    __shared__ scalar_t dcy;    
    __shared__ scalar_t d0x;
    __shared__ scalar_t d0y;
    __shared__ scalar_t dPoint0;
    __shared__ double ang;
    __shared__ double PI;
    __shared__ double ang_error;
    __shared__ double cosval;
    __shared__ double sinval;
    __shared__ unsigned int dIndex0;

    PI = acos(-1.0);
    ang = static_cast<double>(prj_views[idxchannel][idxview]);
    dImage = *dImg;
    ang_error = abs(ang - round(ang / PI) * PI) * 4 / PI;
    cosval = cos(ang);
    sinval = sin(ang);
    sourcex = - sinval * *s2r;
    sourcey = cosval * *s2r;
    dcx = sinval * *d2r + *binshift * cosval;
    dcy = - cosval * *d2r + *binshift * sinval;
    dIndex0 = blockIdx.z * blockDim.x;
    unsigned int tx = threadIdx.x;
    unsigned int dIndex = dIndex0 + tx;
    prj[tx] = 0;
    __syncthreads();
    if (ang_error <= 1) {
        ang_error = (ang - floor(ang / 2 / PI) * 2 * PI) * 4 / PI;
        if (ang_error >= 3 && ang_error < 7) {
            d0x = (*dets / 2 - dIndex0) * *dDet * cosval + dcx;
            d0y = (*dets / 2 - dIndex0) * *dDet * sinval + dcy;
            dPoint0 = map_x(sourcex, sourcey, d0x, d0y);
            if (dIndex < *dets) {
                scalar_t detx = (*dets / 2 - dIndex - 1) * *dDet * cosval + dcx;
                scalar_t dety = (*dets / 2 - dIndex - 1) * *dDet * sinval + dcy;
                dPoint[tx] = map_x(sourcex, sourcey, detx, dety);
            }
        } else {
            d0x = (dIndex0 - *dets / 2) * *dDet * cosval + dcx;
            d0y = (dIndex0 - *dets / 2) * *dDet * sinval + dcy;
            dPoint0 = map_x(sourcex, sourcey, d0x, d0y);
            if (dIndex < *dets) {
                scalar_t detx = (dIndex + 1 - *dets / 2) * *dDet * cosval + dcx;
                scalar_t dety = (dIndex + 1 - *dets / 2) * *dDet * sinval + dcy;
                dPoint[tx] = map_x(sourcex, sourcey, detx, dety);
            }
        }
        __syncthreads();
        if (tx == 0){
            coef[tx] = dPoint[tx] - dPoint0;
        } else {
            coef[tx] = dPoint[tx] - dPoint[tx-1];
        }
        __syncthreads();
        for (int i = 0; i < ceil(*height / blockDim.x); i++){
            int idxrow = i * blockDim.x + tx;
            if (idxrow < *height) {
                scalar_t i0y = (*height / 2 - idxrow - 0.5) * dImage;
                scalar_t i0x = - *width / 2 * dImage; 
                int idx0col = floor(((i0y - sourcey) / (d0y - sourcey) * 
                    (d0x - sourcex) + sourcex - i0x) / dImage);
                idx0col = max(idx0col, 0);
                i0x += idx0col * dImage;
                scalar_t threadprj = 0;
                scalar_t prebound = map_x(sourcex, sourcey, i0x, i0y);
                prebound = max(prebound, dPoint0);
                i0x += dImage;
                scalar_t pixbound = map_x(sourcex, sourcey, i0x, i0y);
                scalar_t detbound = dPoint[0];
                int idxd = 0, idxi = idx0col;
                while (idxi < *width && (idxd + dIndex0) < *dets && idxd < blockDim.x) {
                    if (detbound <= prebound) {
                        idxd ++;                        
                        if (idxd < blockDim.x) detbound = dPoint[idxd];
                    }else if (pixbound <= prebound){
                        idxi ++;
                        i0x += dImage;
                        pixbound = map_x(sourcex, sourcey, i0x, i0y);
                    }else if (pixbound < detbound) {
                        threadprj += (pixbound - prebound) * image[idxchannel][0][idxrow][idxi] / coef[idxd];
                        prebound = pixbound;
                        idxi ++;                        
                        i0x += dImage;
                        pixbound = map_x(sourcex, sourcey, i0x, i0y);                        
                    } else {
                        threadprj += (detbound - prebound) * image[idxchannel][0][idxrow][idxi] / coef[idxd];
                        prebound = detbound;
                        atomicAdd(prj+idxd, threadprj);
                        threadprj = 0;
                        idxd ++;
                        if (idxd < blockDim.x) detbound = dPoint[idxd];
                    }
                }
                if (threadprj != 0) atomicAdd(prj+idxd, threadprj);
            }
        }
        __syncthreads();
        dPoint0 = abs(sourcey) / sqrt((dPoint0 - sourcex) * (dPoint0 - sourcex)  + sourcey * sourcey);
        if (dIndex < *dets) {
            dPoint[tx] = abs(sourcey) / sqrt((dPoint[tx] - sourcex) * (dPoint[tx] - sourcex) + sourcey * sourcey);
            __syncthreads();
            if (tx == 0){
                coef[tx] = (dPoint[tx] + dPoint0) / 2;
            } else {
                coef[tx] = (dPoint[tx] + dPoint[tx-1]) / 2;
            }
            __syncthreads();            
            prj[tx] *= dImage;
            prj[tx] /= coef[tx];
            if (ang_error >= 3 && ang_error < 7) {
                projection[idxchannel][0][idxview][static_cast<unsigned int>(*dets)-1-dIndex] = prj[tx];
            } else {
                projection[idxchannel][0][idxview][dIndex] = prj[tx];
            }
        }
    } else {
        ang_error = (ang - floor(ang / 2 / PI) * 2 * PI) * 4 / PI;
        if (ang_error >= 3 && ang_error < 7) {
            d0x = (*dets / 2 - dIndex0) * *dDet * cosval + dcx;
            d0y = (*dets / 2 - dIndex0) * *dDet * sinval + dcy;
            dPoint0 = map_y(sourcex, sourcey, d0x, d0y);
            if (dIndex < *dets) {
                scalar_t detx = (*dets / 2 - dIndex - 1) * *dDet * cosval + dcx;
                scalar_t dety = (*dets / 2 - dIndex - 1) * *dDet * sinval + dcy;
                dPoint[tx] = map_y(sourcex, sourcey, detx, dety);
            }
        } else {
            d0x = (dIndex0 - *dets / 2) * *dDet * cosval + dcx;
            d0y = (dIndex0 - *dets / 2) * *dDet * sinval + dcy;
            dPoint0 = map_y(sourcex, sourcey, d0x, d0y);
            if (dIndex < *dets) {
                scalar_t detx = (dIndex + 1 - *dets / 2) * *dDet * cosval + dcx;
                scalar_t dety = (dIndex + 1 - *dets / 2) * *dDet * sinval + dcy;
                dPoint[tx] = map_y(sourcex, sourcey, detx, dety);
            }
        }
        __syncthreads();
        if (tx == 0){
            coef[tx] = dPoint[tx] - dPoint0;
        } else {
            coef[tx] = dPoint[tx] - dPoint[tx-1];
        }
        __syncthreads();
        for (int i = 0; i < ceil(*width / blockDim.x); i++){
            int idxcol = i * blockDim.x + tx;
            if (idxcol < *width) {
                scalar_t i0x = (idxcol - *width / 2 + 0.5) * dImage;
                scalar_t i0y = - *height / 2 * dImage; 
                int idx0row = floor(((i0x - sourcex) / (d0x - sourcex) * 
                    (d0y - sourcey) + sourcey - i0y) / dImage);
                idx0row = max(idx0row, 0);
                i0y += idx0row * dImage;
                scalar_t threadprj = 0;
                scalar_t prebound = map_y(sourcex, sourcey, i0x, i0y);
                prebound = max(prebound, dPoint0);
                i0y += dImage;
                scalar_t pixbound = map_y(sourcex, sourcey, i0x, i0y);
                scalar_t detbound = dPoint[0];
                int idxd = 0, idxi = idx0row;
                while (idxi < *height && (idxd + dIndex0) < *dets && idxd < blockDim.x) {
                    if (detbound <= prebound) {
                        idxd ++;
                        if (idxd < blockDim.x) detbound = dPoint[idxd];
                    }else if (pixbound <= prebound) {
                        idxi ++;
                        i0y += dImage;
                        pixbound = map_y(sourcex, sourcey, i0x, i0y);
                    }else if (pixbound < detbound) {
                        threadprj += (pixbound - prebound) * image[idxchannel][0][static_cast<int>(*height)-1-idxi][idxcol] / coef[idxd];
                        prebound = pixbound;
                        idxi ++;
                        i0y += dImage;
                        pixbound = map_y(sourcex, sourcey, i0x, i0y);
                    } else {
                        threadprj += (detbound - prebound) * image[idxchannel][0][static_cast<int>(*height)-1-idxi][idxcol] / coef[idxd];
                        prebound = detbound;
                        atomicAdd(prj+idxd, threadprj);
                        threadprj = 0;
                        idxd ++;
                        if (idxd < blockDim.x) detbound = dPoint[idxd];
                    }
                }
                if (threadprj != 0) atomicAdd(prj+idxd, threadprj);
            }
        }
        __syncthreads();
        dPoint0 = abs(sourcex) / sqrt((dPoint0 - sourcey) * (dPoint0 - sourcey)  + sourcex * sourcex);
        if (dIndex < *dets) {
            dPoint[tx] = abs(sourcex) / sqrt((dPoint[tx] - sourcey) * (dPoint[tx] - sourcey) + sourcex * sourcex);
            __syncthreads();
            if (tx == 0){
                coef[tx] = (dPoint[tx] + dPoint0) / 2;
            } else {
                coef[tx] = (dPoint[tx] + dPoint[tx-1]) / 2;
            }
            __syncthreads();
            prj[tx] *= dImage;
            prj[tx] /= coef[tx];
            if (ang_error >= 3 && ang_error < 7) {
                projection[idxchannel][0][idxview][static_cast<unsigned int>(*dets)-1-dIndex] = prj[tx];
            } else {
                projection[idxchannel][0][idxview][dIndex] = prj[tx];
            }
        }
    }   
}

template <typename scalar_t>
__global__ void bprj_fan_ed(
    torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> image,
    const torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> projection,
    const torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> prj_views,
    const scalar_t* __restrict__ views, const scalar_t* __restrict__ dets,
    const scalar_t* __restrict__ width, const scalar_t* __restrict__ height,
    const scalar_t* __restrict__ dImg, const scalar_t* __restrict__ dDet,
    const scalar_t* __restrict__ dAng, const scalar_t* __restrict__ s2r,
    const scalar_t* __restrict__ d2r, const scalar_t* __restrict__ binshift) {
    
    __shared__ unsigned int nblocks;
    __shared__ unsigned int idxchannel;
    __shared__ unsigned int idxview;
    nblocks = ceil(*views / gridDim.y);
    idxchannel = blockIdx.x % nblocks;
    idxview = idxchannel * gridDim.y + blockIdx.y;
    if (idxview >= *views)   return;
    idxchannel = blockIdx.x / nblocks;
    __shared__ scalar_t prj[BLOCK_DIM];
    __shared__ scalar_t dPoint[BLOCK_DIM];
    __shared__ scalar_t coef[BLOCK_DIM];    
    __shared__ scalar_t dImage;    
    __shared__ scalar_t sourcex;
    __shared__ scalar_t sourcey;
    __shared__ scalar_t dcx;
    __shared__ scalar_t dcy;    
    __shared__ scalar_t d0x;
    __shared__ scalar_t d0y;
    __shared__ scalar_t dPoint0;
    __shared__ double ang;
    __shared__ double PI;
    __shared__ double ang_error;
    __shared__ double cosval;
    __shared__ double sinval;
    __shared__ unsigned int dIndex0;

    PI = acos(-1.0);
    ang = static_cast<double>(prj_views[idxchannel][idxview]);
    ang_error = abs(ang - round(ang / PI) * PI) * 4 / PI;
    cosval = cos(ang);
    sinval = sin(ang);
    sourcex = - sinval * *s2r;
    sourcey = cosval * *s2r;
    dcx = sinval * *d2r + *binshift * cosval;
    dcy = - cosval * *d2r + *binshift * sinval;
    dIndex0 = blockIdx.z * blockDim.x;
    unsigned int tx = threadIdx.x;
    unsigned int dIndex = dIndex0 + tx;
    __syncthreads();
    if (ang_error <= 1) {
        ang_error = (ang - floor(ang / 2 / PI) * 2 * PI) * 4 / PI;
        if (ang_error >= 3 && ang_error < 7) {
            d0x = (*dets / 2 - dIndex0) * *dDet * cosval + dcx;
            d0y = (*dets / 2 - dIndex0) * *dDet * sinval + dcy;
            dPoint0 = map_x(sourcex, sourcey, d0x, d0y);
            if (dIndex < *dets) {
                scalar_t detx = (*dets / 2 - dIndex - 1) * *dDet * cosval + dcx;
                scalar_t dety = (*dets / 2 - dIndex - 1) * *dDet * sinval + dcy;
                dPoint[tx] = map_x(sourcex, sourcey, detx, dety);
            }
        } else {
            d0x = (dIndex0 - *dets / 2) * *dDet * cosval + dcx;
            d0y = (dIndex0 - *dets / 2) * *dDet * sinval + dcy;
            dPoint0 = map_x(sourcex, sourcey, d0x, d0y);
            if (dIndex < *dets) {
                scalar_t detx = (dIndex + 1 - *dets / 2) * *dDet * cosval + dcx;
                scalar_t dety = (dIndex + 1 - *dets / 2) * *dDet * sinval + dcy;
                dPoint[tx] = map_x(sourcex, sourcey, detx, dety);
            }
        }
        __syncthreads();
        dImage = abs(sourcey) / sqrt((dPoint0 - sourcex) * (dPoint0 - sourcex)  + sourcey * sourcey);
        if (dIndex < *dets) {
            prj[tx] = abs(sourcey) / sqrt((dPoint[tx] - sourcex) * (dPoint[tx] - sourcex) + sourcey * sourcey);
            __syncthreads();
            if (tx == 0){
                coef[tx] = (prj[tx] + dImage) / 2;
            } else {
                coef[tx] = (prj[tx] + prj[tx-1]) / 2;
            }
            __syncthreads();
            if (ang_error >= 3 && ang_error < 7) {
                prj[tx] = projection[idxchannel][0][idxview][static_cast<unsigned int>(*dets)-1-dIndex];
            } else {
                prj[tx] = projection[idxchannel][0][idxview][dIndex];
            }
            prj[tx] *= *dImg;
            prj[tx] /= coef[tx];
            if (tx == 0){
                coef[tx] = dPoint[tx] - dPoint0;
            } else {
                coef[tx] = dPoint[tx] - dPoint[tx-1];
            }
        } else {
            prj[tx] = 0;
            coef[tx] = 1;
        }        
        __syncthreads();
        dImage = *dImg;
        for (int i = 0; i < ceil(*height / blockDim.x); i++){
            int idxrow = i * blockDim.x + tx;
            if (idxrow < *height) {
                scalar_t i0y = (*height / 2 - idxrow - 0.5) * dImage;
                scalar_t i0x = - *width / 2 * dImage; 
                int idx0col = floor(((i0y - sourcey) / (d0y - sourcey) * 
                    (d0x - sourcex) + sourcex - i0x) / dImage);
                idx0col = max(idx0col, 0);
                i0x += idx0col * dImage;
                scalar_t threadprj = 0;
                scalar_t prebound = map_x(sourcex, sourcey, i0x, i0y);
                prebound = max(prebound, dPoint0);
                i0x += dImage;
                scalar_t pixbound = map_x(sourcex, sourcey, i0x, i0y);
                scalar_t detbound = dPoint[0];
                int idxd = 0, idxi = idx0col;
                while (idxi < *width && (idxd + dIndex0) < *dets && idxd < blockDim.x) {
                    if (detbound <= prebound) {
                        idxd ++;                        
                        if (idxd < blockDim.x) detbound = dPoint[idxd];
                    }else if (pixbound <= prebound){
                        idxi ++;
                        i0x += dImage;
                        pixbound = map_x(sourcex, sourcey, i0x, i0y);
                    }else if (pixbound <= detbound) {
                        threadprj += (pixbound - prebound) * prj[idxd] / coef[idxd];
                        prebound = pixbound;
                        atomicAdd(&(image[idxchannel][0][idxrow][idxi]), threadprj);
                        threadprj = 0;
                        idxi ++;
                        i0x += dImage;
                        pixbound = map_x(sourcex, sourcey, i0x, i0y);
                    } else {
                        threadprj += (detbound - prebound) * prj[idxd] / coef[idxd];
                        prebound = detbound;
                        idxd ++;
                        if (idxd < blockDim.x) detbound = dPoint[idxd];
                    }
                }
                if (threadprj !=0 ) atomicAdd(&(image[idxchannel][0][idxrow][idxi]), threadprj);
            }
        }
    } else {
        ang_error = (ang - floor(ang / 2 / PI) * 2 * PI) * 4 / PI;
        if (ang_error >= 3 && ang_error < 7) {
            d0x = (*dets / 2 - dIndex0) * *dDet * cosval + dcx;
            d0y = (*dets / 2 - dIndex0) * *dDet * sinval + dcy;
            dPoint0 = map_y(sourcex, sourcey, d0x, d0y);
            if (dIndex < *dets) {
                scalar_t detx = (*dets / 2 - dIndex - 1) * *dDet * cosval + dcx;
                scalar_t dety = (*dets / 2 - dIndex - 1) * *dDet * sinval + dcy;
                dPoint[tx] = map_y(sourcex, sourcey, detx, dety);
            }
        } else {
            d0x = (dIndex0 - *dets / 2) * *dDet * cosval + dcx;
            d0y = (dIndex0 - *dets / 2) * *dDet * sinval + dcy;
            dPoint0 = map_y(sourcex, sourcey, d0x, d0y);
            if (dIndex < *dets) {
                scalar_t detx = (dIndex + 1 - *dets / 2) * *dDet * cosval + dcx;
                scalar_t dety = (dIndex + 1 - *dets / 2) * *dDet * sinval + dcy;
                dPoint[tx] = map_y(sourcex, sourcey, detx, dety);
            }
        }
        __syncthreads();
        dImage = abs(sourcex) / sqrt((dPoint0 - sourcey) * (dPoint0 - sourcey)  + sourcex * sourcex);
        if (dIndex < *dets) {
            prj[tx] = abs(sourcex) / sqrt((dPoint[tx] - sourcey) * (dPoint[tx] - sourcey) + sourcex * sourcex);
            __syncthreads();
            if (tx == 0){
                coef[tx] = (prj[tx] + dImage) / 2;
            } else {
                coef[tx] = (prj[tx] + prj[tx-1]) / 2;
            }
            __syncthreads();
            if (ang_error >= 3 && ang_error < 7) {
                prj[tx] = projection[idxchannel][0][idxview][static_cast<unsigned int>(*dets)-1-dIndex];
            } else {
                prj[tx] = projection[idxchannel][0][idxview][dIndex];
            }
            prj[tx] *= *dImg;
            prj[tx] /= coef[tx];
            if (tx == 0){
                coef[tx] = dPoint[tx] - dPoint0;
            } else {
                coef[tx] = dPoint[tx] - dPoint[tx-1];
            }
        } else {
            prj[tx] = 0;
            coef[tx] = 1;
        }        
        __syncthreads();
        dImage = *dImg;
        for (int i = 0; i < ceil(*width / blockDim.x); i++){
            int idxcol = i * blockDim.x + tx;
            if (idxcol < *width) {
                scalar_t i0x = (idxcol - *width / 2 + 0.5) * dImage;
                scalar_t i0y = - *height / 2 * dImage; 
                int idx0row = floor(((i0x - sourcex) / (d0x - sourcex) * 
                    (d0y - sourcey) + sourcey - i0y) / dImage);
                idx0row = max(idx0row, 0);
                i0y += idx0row * dImage;
                scalar_t threadprj = 0;
                scalar_t prebound = map_y(sourcex, sourcey, i0x, i0y);
                prebound = max(prebound, dPoint0);
                i0y += dImage;
                scalar_t pixbound = map_y(sourcex, sourcey, i0x, i0y);
                scalar_t detbound = dPoint[0];
                int idxd = 0, idxi = idx0row;
                while (idxi < *height && (idxd + dIndex0) < *dets && idxd < blockDim.x) {
                    if (detbound <= prebound) {
                        idxd ++;
                        if (idxd < blockDim.x) detbound = dPoint[idxd];
                    }else if (pixbound <= prebound) {
                        idxi ++;
                        i0y += dImage;
                        pixbound = map_y(sourcex, sourcey, i0x, i0y);
                    }else if (pixbound <= detbound) {
                        threadprj += (pixbound - prebound) * prj[idxd] / coef[idxd];
                        prebound = pixbound;
                        atomicAdd(&(image[idxchannel][0][static_cast<int>(*height)-1-idxi][idxcol]), threadprj);
                        threadprj = 0;
                        idxi ++;
                        i0y += dImage;
                        pixbound = map_y(sourcex, sourcey, i0x, i0y);
                    } else {
                        threadprj += (detbound - prebound) * prj[idxd] / coef[idxd];
                        prebound = detbound;                        
                        idxd ++;
                        if (idxd < blockDim.x) detbound = dPoint[idxd];
                    }
                }
                if (threadprj !=0 ) atomicAdd(&(image[idxchannel][0][static_cast<int>(*height)-1-idxi][idxcol]), threadprj);
            }
        }
    }   
}

torch::Tensor prj_fan_ed_view_cuda(torch::Tensor image, torch::Tensor options, torch::Tensor prj_views) {
    hipSetDevice(image.device().index());
    auto views = options[0];
    auto dets = options[1];
    auto width = options[2];
    auto height = options[3];
    auto dImg = options[4];
    auto dDet = options[5];
    auto dAng = options[6];
    auto s2r = options[7];
    auto d2r = options[8];
    auto binshift = options[9];
    const int channels = static_cast<int>(image.size(0));
    auto projection = torch::empty({channels, 1, views.item<int>(), dets.item<int>()}, image.options());
    
    int nblocksx = ceil(views.item<float>() / GRID_DIM) * channels;
    int nblocksy = min(views.item<int>(), GRID_DIM);
    int nblocksz = ceil(dets.item<float>() / BLOCK_DIM);
    const dim3 blocks(nblocksx, nblocksy, nblocksz);

    AT_DISPATCH_FLOATING_TYPES(image.type(), "fan_beam_equal_distance_projection", ([&] {
        prj_fan_ed<scalar_t><<<blocks, BLOCK_DIM>>>(
            image.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
            projection.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
            prj_views.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            views.data<scalar_t>(), dets.data<scalar_t>(), width.data<scalar_t>(),
            height.data<scalar_t>(), dImg.data<scalar_t>(), dDet.data<scalar_t>(),
            dAng.data<scalar_t>(), s2r.data<scalar_t>(), d2r.data<scalar_t>(),
            binshift.data<scalar_t>()
        );
    }));
    return projection;
}

torch::Tensor bprj_fan_ed_view_cuda(torch::Tensor projection, torch::Tensor options, torch::Tensor prj_views) {
    hipSetDevice(projection.device().index());
    auto views = options[0];
    auto dets = options[1];
    auto width = options[2];
    auto height = options[3];
    auto dImg = options[4];
    auto dDet = options[5];
    auto dAng = options[6];
    auto s2r = options[7];
    auto d2r = options[8];
    auto binshift = options[9];
    const int channels = static_cast<int>(projection.size(0));
    auto image = torch::zeros({channels, 1, height.item<int>(), width.item<int>()}, projection.options());
    
    int nblocksx = ceil(views.item<float>() / GRID_DIM) * channels;
    int nblocksy = min(views.item<int>(), GRID_DIM);
    int nblocksz = ceil(dets.item<float>() / BLOCK_DIM);
    const dim3 blocks(nblocksx, nblocksy, nblocksz);

    AT_DISPATCH_FLOATING_TYPES(projection.type(), "fan_beam_equal_distance_backprojection", ([&] {
        bprj_fan_ed<scalar_t><<<blocks, BLOCK_DIM>>>(
            image.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
            projection.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
            prj_views.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
            views.data<scalar_t>(), dets.data<scalar_t>(), width.data<scalar_t>(),
            height.data<scalar_t>(), dImg.data<scalar_t>(), dDet.data<scalar_t>(),
            dAng.data<scalar_t>(), s2r.data<scalar_t>(), d2r.data<scalar_t>(),
            binshift.data<scalar_t>()
        );
    }));
    return image;
}